#include "hip/hip_runtime.h"
// Adapted from https://github.com/vllm-project/vllm/blob/v0.6.5/csrc/moe/moe_align_sum_kernels.cu

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <THC/THCAtomics.cuh>

#include "utils.hpp"

#ifdef USE_ROCM
#include <hip/hip_runtime.h>
#endif

#ifndef USE_ROCM
#define WARP_SIZE 32
#else
#define WARP_SIZE warpSize
#endif

#ifndef USE_ROCM
#define DevFuncAttribute_SET_MaxDynamicSharedMemorySize(FUNC, VAL) \
  hipFuncSetAttribute(reinterpret_cast<const void*>(FUNC), hipFuncAttributeMaxDynamicSharedMemorySize, VAL)
#else
#define DevFuncAttribute_SET_MaxDynamicSharedMemorySize(FUNC, VAL) \
  hipFuncSetAttribute(reinterpret_cast<const void*>(FUNC), hipFuncAttributeMaxDynamicSharedMemorySize, VAL)
#endif

#define CEILDIV(x, y) (((x) + (y)-1) / (y))

#define DISPATCH_CASE_INTEGRAL_TYPES(...)              \
  AT_DISPATCH_CASE(at::ScalarType::Byte, __VA_ARGS__)  \
  AT_DISPATCH_CASE(at::ScalarType::Char, __VA_ARGS__)  \
  AT_DISPATCH_CASE(at::ScalarType::Short, __VA_ARGS__) \
  AT_DISPATCH_CASE(at::ScalarType::Int, __VA_ARGS__)   \
  AT_DISPATCH_CASE(at::ScalarType::Long, __VA_ARGS__)

#define DISPATCH_INTEGRAL_TYPES(TYPE, NAME, ...) \
  AT_DISPATCH_SWITCH(TYPE, NAME, DISPATCH_CASE_INTEGRAL_TYPES(__VA_ARGS__))

__device__ __forceinline__ int32_t index(int32_t total_col, int32_t row, int32_t col) {
  // don't worry about overflow because num_experts is relatively small
  return row * total_col + col;
}

template <typename scalar_t>
__global__ void moe_align_block_size_kernel(scalar_t* __restrict__ topk_ids, int32_t* sorted_token_ids,
                                            int32_t* expert_ids, int32_t* total_tokens_post_pad, int32_t num_experts,
                                            int32_t block_size, size_t numel) {
  __shared__ int32_t shared_counts[32][8];
  __shared__ int32_t local_offsets[256];
  __shared__ int32_t shared_cumsum[257];

  const int warp_id = threadIdx.x / WARP_SIZE;
  const int experts_per_warp = 8;
  const int my_expert_start = warp_id * experts_per_warp;

  for (int i = 0; i < experts_per_warp; ++i) {
    if (my_expert_start + i < num_experts) {
      shared_counts[warp_id][i] = 0;
    }
  }

  const size_t tokens_per_thread = CEILDIV(numel, blockDim.x);
  const size_t start_idx = threadIdx.x * tokens_per_thread;

  for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
    int expert_id = topk_ids[i];
    int warp_idx = expert_id / experts_per_warp;
    int expert_offset = expert_id % experts_per_warp;
    atomicAdd(&shared_counts[warp_idx][expert_offset], 1);
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    shared_cumsum[0] = 0;
    for (int i = 1; i <= num_experts; ++i) {
      int expert_count = 0;
      int warp_idx = (i - 1) / experts_per_warp;
      int expert_offset = (i - 1) % experts_per_warp;
      expert_count = shared_counts[warp_idx][expert_offset];

      shared_cumsum[i] = shared_cumsum[i - 1] + CEILDIV(expert_count, block_size) * block_size;
    }
    *total_tokens_post_pad = shared_cumsum[num_experts];
  }

  __syncthreads();

  if (threadIdx.x < num_experts) {
    for (int i = shared_cumsum[threadIdx.x]; i < shared_cumsum[threadIdx.x + 1]; i += block_size) {
      expert_ids[i / block_size] = threadIdx.x;
    }
    local_offsets[threadIdx.x] = shared_cumsum[threadIdx.x];
  }

  __syncthreads();

  for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
    int32_t expert_id = topk_ids[i];
    int32_t rank_post_pad = atomicAdd(&local_offsets[expert_id], 1);
    sorted_token_ids[rank_post_pad] = i;
  }
}

void moe_align_block_size(torch::Tensor topk_ids, int64_t num_experts, int64_t block_size,
                          torch::Tensor sorted_token_ids, torch::Tensor experts_ids, torch::Tensor num_tokens_post_pad) {
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  const int threads_per_block = 1024;
  assert(num_experts == 256 && "num_experts must be 256 now.");
  DISPATCH_INTEGRAL_TYPES(topk_ids.scalar_type(), "moe_align_block_size_kernel", [&] {
    auto kernel = moe_align_block_size_kernel<scalar_t>;
    kernel<<<1, threads_per_block, 0, stream>>>(topk_ids.data_ptr<scalar_t>(), sorted_token_ids.data_ptr<int32_t>(),
                                   experts_ids.data_ptr<int32_t>(), num_tokens_post_pad.data_ptr<int32_t>(),
                                   num_experts, block_size, topk_ids.numel());
  });
}
