#include "hip/hip_runtime.h"
/* Copyright 2025 SGLang Team. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "act_and_mul_internal.cuh"

namespace flashinfer {
namespace activation {

template <typename T>
__device__ __forceinline__ T gelu_tanh(const T& x) {
  constexpr float kAlpha = 0.044715f;
  constexpr float kBeta = 0.7978845608028654f;
  const float f32_val = castToFloat(x);

  const float f32_val_pow_of_3 = __powf(f32_val, 3.f);
  const float cdf =
    0.5f * (1.0f + tanhf( ( kBeta * ( f32_val + kAlpha  * f32_val_pow_of_3 ) ) ) );

  return castFrom<T>( f32_val * cdf );
}

} // activation
} // flashinfer


void gelu_tanh_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream) {
  int d = input.size(-1) / 2;
  int64_t num_tokens = input.numel() / input.size(-1);
  dim3 grid(num_tokens);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), c_type, [&] {
    uint32_t vec_size = 16 / sizeof(c_type);
    dim3 block(std::min(d / vec_size, 1024U));

    flashinfer::activation::act_and_mul_kernel<c_type, flashinfer::activation::gelu_tanh><<<grid, block, 0, stream>>>(
        static_cast<c_type*>(out.data_ptr()), static_cast<c_type*>(input.data_ptr()), d);

    return true;
  });
}
