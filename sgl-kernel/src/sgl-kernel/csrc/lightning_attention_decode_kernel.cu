#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "utils.hpp"

#define THREADS_PER_BLOCK 128

template<typename T>
__global__ void lightning_attention_decode_kernel(
    const T* __restrict__ q,      // [b, h, 1, d]
    const T* __restrict__ k,      // [b, h, 1, d]
    const T* __restrict__ v,      // [b, h, 1, e]
    const float* __restrict__ past_kv, // [b, h, d, e]
    const float* __restrict__ slope,   // [h, 1, 1]
    T* __restrict__ output,       // [b, h, 1, e]
    float* __restrict__ new_kv,   // [b, h, d, e]
    const int batch_size,
    const int num_heads,
    const int dim,
    const int embed_dim) {
    
    extern __shared__ char smem[];
    T* q_shared = reinterpret_cast<T*>(smem);
    T* k_shared = reinterpret_cast<T*>(smem + dim * sizeof(T));
    T* v_shared = reinterpret_cast<T*>(smem + 2 * dim * sizeof(T));
    float* new_kv_shared = reinterpret_cast<float*>(smem + (2 * dim + embed_dim) * sizeof(T));
    T* output_shared = reinterpret_cast<T*>(smem + (2 * dim + embed_dim) * sizeof(T) + dim * (embed_dim + 1) * sizeof(float));
    
    const int32_t tid = threadIdx.x;
    const int32_t current_head = blockIdx.x;
    const int32_t b = current_head / num_heads;
    const int32_t h = current_head % num_heads;
    
    if (b >= batch_size) return;
    
    const int32_t qk_offset = b * num_heads * dim + h * dim;
    const int32_t v_offset = b * num_heads * embed_dim + h * embed_dim;
    const int32_t kv_offset = b * num_heads * dim * embed_dim + h * dim * embed_dim;
    
    for (int d = tid; d < dim; d += blockDim.x) {
        q_shared[d] = q[qk_offset + d];
        k_shared[d] = k[qk_offset + d];
    }
    for (int e = tid; e < embed_dim; e += blockDim.x) {
        v_shared[e] = v[v_offset + e];
    }
    
    __syncthreads();
    
    const float ratio = expf(-1.0f * slope[h]);
    
    for (int d = tid; d < dim; d += blockDim.x) {
        T k_val = k_shared[d];
        for (int e = 0; e < embed_dim; ++e) {
            int past_kv_idx = kv_offset + d * embed_dim + e;
            T v_val = v_shared[e];
            float new_val = ratio * past_kv[past_kv_idx] + k_val * v_val;
            int shared_idx = d * (embed_dim + 1) + e;
            new_kv_shared[shared_idx] = new_val;
        }
    }
    
    __syncthreads();
    
    for (int idx = tid; idx < dim * embed_dim; idx += blockDim.x) {
        int d = idx / embed_dim;
        int e = idx % embed_dim;
        int shared_idx = d * (embed_dim + 1) + e;
        int global_idx = kv_offset + idx;
        new_kv[global_idx] = new_kv_shared[shared_idx];
    }
    
    __syncthreads();
    
    for (int e = tid; e < embed_dim; e += blockDim.x) {
        float sum = 0.0f;
        for (int d = 0; d < dim; ++d) {
            int shared_idx = d * (embed_dim + 1) + e;
            sum += q_shared[d] * new_kv_shared[shared_idx];
        }
        output_shared[e] = static_cast<T>(sum);
    }
    
    __syncthreads();
    
    if (tid == 0) {
        for (int e = 0; e < embed_dim; ++e) {
            output[v_offset + e] = output_shared[e];
        }
    }
}

void lightning_attention_decode(
    const torch::Tensor& q,
    const torch::Tensor& k, 
    const torch::Tensor& v,
    const torch::Tensor& past_kv,
    const torch::Tensor& slope,
    torch::Tensor output,
    torch::Tensor new_kv) {
    
    TORCH_CHECK(q.is_contiguous(), "q must be contiguous");
    TORCH_CHECK(k.is_contiguous(), "k must be contiguous");
    TORCH_CHECK(v.is_contiguous(), "v must be contiguous");
    TORCH_CHECK(past_kv.is_contiguous(), "past_kv must be contiguous");
    
    auto batch_size = q.size(0);
    auto num_heads = q.size(1);
    auto dim = q.size(3);
    auto embed_dim = v.size(3);
    
    dim3 block(THREADS_PER_BLOCK);
    dim3 grid(batch_size * num_heads);
    
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16, q.scalar_type(), "lightning_attention_decode_kernel", ([&] {
        size_t smem_size = (2 * dim + 2 * embed_dim) * sizeof(scalar_t) + dim * (embed_dim + 1) * sizeof(float);
        lightning_attention_decode_kernel<scalar_t><<<grid, block, smem_size, stream>>>(
            q.data_ptr<scalar_t>(),
            k.data_ptr<scalar_t>(),
            v.data_ptr<scalar_t>(),
            past_kv.data_ptr<float>(),
            slope.data_ptr<float>(),
            output.data_ptr<scalar_t>(),
            new_kv.data_ptr<float>(),
            batch_size,
            num_heads,
            dim,
            embed_dim
        );
    }));
}