#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "utils.hpp"

#define WARP_SIZE 32
#define WARPS_PER_BLOCK 4

template<typename T>
__global__ void lightning_attention_decode_kernel(
    const T* __restrict__ q,      // [b, h, 1, d]
    const T* __restrict__ k,      // [b, h, 1, d]
    const T* __restrict__ v,      // [b, h, 1, e]
    const float* __restrict__ past_kv,// [b, h, d, e]
    const float* __restrict__ slope,  // [h, 1, 1]
    T* __restrict__ output,       // [b, h, 1, e]
    float* __restrict__ new_kv,   // [b, h, d, e]
    const int batch_size,
    const int num_heads,
    const int qk_dim,
    const int v_dim) {
    
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int lane_id = tx;
    
    const int current_head = blockDim.y * blockIdx.x + ty;
    const int b = current_head / num_heads;
    const int h = current_head % num_heads;
    
    if (b >= batch_size) return;
    
    const int64_t qk_offset = b * num_heads * qk_dim + h * qk_dim;
    const int64_t v_offset = b * num_heads * v_dim + h * v_dim;
    const int64_t kv_offset = b * num_heads * qk_dim * v_dim + h * qk_dim * v_dim;
    
    // 1. Calculate new kv: kv = ratio * kv + k * v^T
    const float ratio = exp(-1.0f * slope[h]);
    for (int d = lane_id; d < qk_dim; d += WARP_SIZE) {
        for (int e = 0; e < v_dim; e++) {
            float val = ratio * past_kv[kv_offset + d * v_dim + e];
            val = val + k[qk_offset + d] * v[v_offset + e];
            new_kv[kv_offset + d * v_dim + e] = val;
        }
    }
    
    // 2. Calculate qkv attention output: output = q * kv
    for (int e = lane_id; e < v_dim; e += WARP_SIZE) {
        float sum = 0.0f;
        for (int d = 0; d < qk_dim; d++) {
            sum += q[qk_offset + d] * 
                  new_kv[kv_offset + d * v_dim + e];
        }
        output[v_offset + e] = static_cast<T>(sum);
    }
}

void lightning_attention_decode(
    const torch::Tensor& q,
    const torch::Tensor& k, 
    const torch::Tensor& v,
    const torch::Tensor& past_kv,
    const torch::Tensor& slope,
    torch::Tensor output,
    torch::Tensor new_kv) {
    
    TORCH_CHECK(q.is_contiguous(), "q must be contiguous");
    TORCH_CHECK(k.is_contiguous(), "k must be contiguous");
    TORCH_CHECK(v.is_contiguous(), "v must be contiguous");
    TORCH_CHECK(past_kv.is_contiguous(), "past_kv must be contiguous");
    
    auto batch_size = q.size(0);
    auto num_heads = q.size(1);
    auto qk_dim = q.size(3);
    auto v_dim = v.size(3);
    
    dim3 block(WARP_SIZE, WARPS_PER_BLOCK);  // (32, 4)
    dim3 grid((batch_size * num_heads + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK);
    
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16, q.scalar_type(), "lightning_attention_decode_kernel", ([&] {
        lightning_attention_decode_kernel<scalar_t><<<grid, block, 0, stream>>>(
            q.data_ptr<scalar_t>(),
            k.data_ptr<scalar_t>(),
            v.data_ptr<scalar_t>(),
            past_kv.data_ptr<float>(),
            slope.data_ptr<float>(),
            output.data_ptr<scalar_t>(),
            new_kv.data_ptr<float>(),
            batch_size,
            num_heads,
            qk_dim,
            v_dim
        );
    }));
}