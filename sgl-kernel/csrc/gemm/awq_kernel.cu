#include "hip/hip_runtime.h"
// Adapted from
// https://github.com/vllm-project/vllm/blob/eb59b5a6cba6727d3727c0372258db9002f687c1/csrc/quantization/awq/gemm_kernels.cu#L350
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_fp16.h>
#include <torch/all.h>

template<bool norm = true>
inline __device__ uint4 dequantize_s4_to_bf16x2(uint32_t const& source) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
  uint4 result;

  uint32_t* h = reinterpret_cast<uint32_t*>(&result);
  uint32_t const& i4s = reinterpret_cast<uint32_t const&>(source);

  // 01234567 01234567
  // SEEEEEEE EMMMMMMM
  // 127 + 7 = 134 -> 0100 0011 0 -> 0x43
  static constexpr uint32_t immLut   = (0xf0 & 0xcc) | 0xaa;
  static constexpr uint32_t BOTTOM_MASK = 0x000f000f;
  static constexpr uint32_t TOP_MASK = 0x00f000f0;
  static constexpr uint32_t I4s_TO_BF16s_MAGIC_NUM = 0x43004300;

  const uint32_t  top_i4s = i4s >> 8;

  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[0])
               : "r"(i4s), "n"(BOTTOM_MASK), "n"(I4s_TO_BF16s_MAGIC_NUM), "n"(immLut));
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[1])
               : "r"(i4s), "n"(TOP_MASK), "n"(I4s_TO_BF16s_MAGIC_NUM), "n"(immLut));
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[2])
               : "r"(top_i4s), "n"(BOTTOM_MASK), "n"(I4s_TO_BF16s_MAGIC_NUM), "n"(immLut));
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[3])
               : "r"(top_i4s), "n"(TOP_MASK), "n"(I4s_TO_BF16s_MAGIC_NUM), "n"(immLut));

  // For bottom 4 bits, need to subtract 2 ^ 7 = 128
  static constexpr uint32_t BF16_TOP_MAGIC_NUM = 0x43004300;
  // For top 4 bits, need to move 4 bits to the right, 2 ^ 4 = 16
  // This is the bf16x2 {1 / 16, 1 / 16} represented as an integer.
  // 0011 1101 1000 0000
  static constexpr uint32_t ONE_SIXTEENTH = 0x3d803d80;
  // 128 / 16 = 8 we need to subtract 8 from the above.
  // This is the bf16x2 {-8, -8} represented as an integer.
  static constexpr uint32_t NEG_8 = 0xc100c100;

  // Convert elt_01
  asm volatile("sub.bf16x2 %0, %1, %2;\n" : "=r"(h[0]) : "r"(h[0]), "r"(BF16_TOP_MAGIC_NUM));
  // Convert elt_23
  asm volatile("fma.rn.bf16x2 %0, %1, %2, %3;\n" : "=r"(h[1]) : "r"(h[1]), "r"(ONE_SIXTEENTH), "r"(NEG_8));
  // Convert elt_45
  asm volatile("sub.bf16x2 %0, %1, %2;\n" : "=r"(h[2]) : "r"(h[2]), "r"(BF16_TOP_MAGIC_NUM));
  // Convert elt_67
  asm volatile("fma.rn.bf16x2 %0, %1, %2, %3;\n" : "=r"(h[3]) : "r"(h[3]), "r"(ONE_SIXTEENTH), "r"(NEG_8));

  return result;
#else
  assert(false);
  return {};
#endif
}

__device__ uint4 dequantize_s4_to_fp16x2(uint32_t const& source) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 750
  uint4 result;

  uint32_t* h = reinterpret_cast<uint32_t*>(&result);
  uint32_t const i4s = reinterpret_cast<uint32_t const&>(source);

  // 01234567 01234567
  // SEEEEEMM MMMMMMMM
  // First, we extract the i4s and construct an intermediate fp16 number.
  static constexpr uint32_t immLut = (0xf0 & 0xcc) | 0xaa;
  static constexpr uint32_t BOTTOM_MASK = 0x000f000f;
  static constexpr uint32_t TOP_MASK = 0x00f000f0;
  static constexpr uint32_t I4s_TO_F16s_MAGIC_NUM = 0x64006400;

  // Note that the entire sequence only requires 1 shift instruction. This is
  // thanks to the register packing format and the fact that we force our
  // integers to be unsigned, and account for this in the fp16 subtractions. In
  // addition, I exploit the fact that sub and fma have the same throughput in
  // order to convert elt_23 and elt_67 to fp16 without having to shift them to
  // the bottom bits before hand.

  // Shift right by 8 to now consider elt_45 and elt_67. Issue first to hide RAW
  // dependency if we issue immediately before required.
  const uint32_t top_i4s = i4s >> 8;
  // Extract elt_01 - (i4s & 0x000f000f) | 0x64006400
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[0])
               : "r"(i4s), "n"(BOTTOM_MASK), "n"(I4s_TO_F16s_MAGIC_NUM), "n"(immLut));
  // Extract elt_23 (i4s & 0x00f000f0) | 0x64006400
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[1])
               : "r"(i4s), "n"(TOP_MASK), "n"(I4s_TO_F16s_MAGIC_NUM), "n"(immLut));
  // Extract elt_45 (top_i4s & 0x000f000f) | 0x64006400
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[2])
               : "r"(top_i4s), "n"(BOTTOM_MASK), "n"(I4s_TO_F16s_MAGIC_NUM), "n"(immLut));
  // Extract elt_67 (top_i4s & 0x00f000f0) | 0x64006400
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[3])
               : "r"(top_i4s), "n"(TOP_MASK), "n"(I4s_TO_F16s_MAGIC_NUM), "n"(immLut));

  // This is the half2 {1024, 1024} represented as an integer.
  // 01234567 01234567
  // SEEEEEMM MMMMMMMM
  static constexpr uint32_t FP16_TOP_MAGIC_NUM = 0x64006400;
  // This is the half2 {1 / 16, 1 / 16} represented as an integer.
  static constexpr uint32_t ONE_SIXTEENTH = 0x2c002c00;
  // 1024 / 16 = 64 we need to subtract 64 from the above.
  // This is the half2 {-64, -64} represented as an integer.
  static constexpr uint32_t NEG_64 = 0xd400d400;

  // Finally, we construct the output numbers.
  // Convert elt_01
  asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h[0]) : "r"(h[0]), "r"(FP16_TOP_MAGIC_NUM));
  // Convert elt_23
  asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(h[1]) : "r"(h[1]), "r"(ONE_SIXTEENTH), "r"(NEG_64));
  // Convert elt_45
  asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h[2]) : "r"(h[2]), "r"(FP16_TOP_MAGIC_NUM));
  // Convert elt_67
  asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(h[3]) : "r"(h[3]), "r"(ONE_SIXTEENTH), "r"(NEG_64));

  return result;
#else
  assert(false);
  return {};
#endif
}

__global__ void __launch_bounds__(256) dequantize_weights(
    int* __restrict__ qweight,
    hip_bfloat16* __restrict__ scales,
    int* __restrict__ qzeros,
    hip_bfloat16* __restrict__ output,
    int group_size,
    int qweight_cols) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  uint4 zeros = dequantize_s4_to_bf16x2(qzeros[col + (row / group_size) * qweight_cols]);
  // uint4 zeros = dequantize_s4_to_fp16x2(qzeros[col + (row / group_size) * qweight_cols]);
  uint4 loaded_scale = *(uint4*)(scales + 8 * col + (row / group_size) * qweight_cols * 8);

  uint4 weight_fp16 = dequantize_s4_to_bf16x2(qweight[col + row * qweight_cols]);
  // uint4 weight_fp16 = dequantize_s4_to_fp16x2(qweight[col + row * qweight_cols]);

  // asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.x) : "r"(weight_fp16.x), "r"(zeros.x));
  // asm volatile("mul.rn.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.x) : "r"(weight_fp16.x), "r"(loaded_scale.x));
  // asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.y) : "r"(weight_fp16.y), "r"(zeros.y));
  // asm volatile("mul.rn.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.y) : "r"(weight_fp16.y), "r"(loaded_scale.y));
  // asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.z) : "r"(weight_fp16.z), "r"(zeros.z));
  // asm volatile("mul.rn.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.z) : "r"(weight_fp16.z), "r"(loaded_scale.z));
  // asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.w) : "r"(weight_fp16.w), "r"(zeros.w));
  // asm volatile("mul.rn.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.w) : "r"(weight_fp16.w), "r"(loaded_scale.w));
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
  asm volatile("sub.bf16x2 %0, %1, %2;\n" : "=r"(weight_fp16.x) : "r"(weight_fp16.x), "r"(zeros.x));
  asm volatile("mul.rn.bf16x2 %0, %1, %2;\n" : "=r"(weight_fp16.x) : "r"(weight_fp16.x), "r"(loaded_scale.x));
  asm volatile("sub.bf16x2 %0, %1, %2;\n" : "=r"(weight_fp16.y) : "r"(weight_fp16.y), "r"(zeros.y));
  asm volatile("mul.rn.bf16x2 %0, %1, %2;\n" : "=r"(weight_fp16.y) : "r"(weight_fp16.y), "r"(loaded_scale.y));
  asm volatile("sub.bf16x2 %0, %1, %2;\n" : "=r"(weight_fp16.z) : "r"(weight_fp16.z), "r"(zeros.z));
  asm volatile("mul.rn.bf16x2 %0, %1, %2;\n" : "=r"(weight_fp16.z) : "r"(weight_fp16.z), "r"(loaded_scale.z));
  asm volatile("sub.bf16x2 %0, %1, %2;\n" : "=r"(weight_fp16.w) : "r"(weight_fp16.w), "r"(zeros.w));
  asm volatile("mul.rn.bf16x2 %0, %1, %2;\n" : "=r"(weight_fp16.w) : "r"(weight_fp16.w), "r"(loaded_scale.w));
#endif
  hip_bfloat16* output_ptr = output + 8 * col + 8 * row * qweight_cols;
  *(uint4*)output_ptr = weight_fp16;
}

torch::Tensor awq_dequantize(torch::Tensor qweight, torch::Tensor scales, torch::Tensor qzeros) {
  int qweight_rows = qweight.size(0);
  int qweight_cols = qweight.size(1);
  int group_size = qweight_rows / scales.size(0);

  int x_num_threads = 16;
  int y_num_threads = 16;
  int x_blocks = qweight_cols / x_num_threads;
  int y_blocks = qweight_rows / y_num_threads;

  const at::cuda::OptionalCUDAGuard device_guard(device_of(qweight));

  auto output_tensor_options = torch::TensorOptions().dtype(scales.dtype()).device(scales.device());
  at::Tensor output = torch::empty({qweight_rows, qweight_cols * 8}, output_tensor_options);

  auto _qweight = reinterpret_cast<int*>(qweight.data_ptr<int>());
  auto _scales = reinterpret_cast<__hip_bfloat16*>(scales.data_ptr<at::BFloat16>());
  auto _zeros = reinterpret_cast<int*>(qzeros.data_ptr<int>());
  auto _output = reinterpret_cast<__hip_bfloat16*>(output.data_ptr<at::BFloat16>());

  dim3 num_blocks(x_blocks, y_blocks);
  dim3 threads_per_block(x_num_threads, y_num_threads);

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  dequantize_weights<<<num_blocks, threads_per_block, 0, stream>>>(
      _qweight, _scales, _zeros, _output, group_size, qweight_cols);

  return output;
}
