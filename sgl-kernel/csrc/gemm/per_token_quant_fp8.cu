#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>

#include <cmath>
#include <cub/block/block_reduce.cuh>
#include <flashinfer/vec_dtypes.cuh>

#include "utils.h"

template <typename T>
__global__ void per_token_quant_fp8_kernel(
    const T* __restrict__ input,
    FP8_TYPE* __restrict__ output_q,
    float* __restrict__ output_s,
    const int64_t hidden_dim,
    const int64_t num_tokens) {
  const int token_idx = blockIdx.x;
  if (token_idx >= num_tokens) return;

  const int tid = threadIdx.x;
  const int block_dim = blockDim.x;

  const T* token_input = input + token_idx * hidden_dim;
  FP8_TYPE* token_output = output_q + token_idx * hidden_dim;

  float max_value = 0.0f;
  constexpr uint32_t vec_size = 16 / sizeof(T);
  const bool use_vector = (hidden_dim % vec_size == 0);

  if (use_vector) {
    using vec_t = flashinfer::vec_t<T, vec_size>;
    const int32_t num_vec_elems = hidden_dim / vec_size;

    for (int32_t i = tid; i < num_vec_elems; i += block_dim) {
      vec_t input_vec;
      input_vec.cast_load(token_input + i * vec_size);

#pragma unroll
      for (uint32_t j = 0; j < vec_size; ++j) {
        max_value = fmaxf(max_value, fabsf(static_cast<float>(input_vec[j])));
      }
    }
  } else {
    for (int32_t idx = tid; idx < hidden_dim; idx += block_dim) {
      max_value = fmaxf(max_value, fabsf(static_cast<float>(token_input[idx])));
    }
  }

  max_value = blockReduceMax(max_value);
  __shared__ float scale_val;
  if (tid == 0) {
    float block_max = max_value / FP8_E4M3_MAX;
    output_s[token_idx] = block_max;
    scale_val = 1.0f / block_max;
  }
  __syncthreads();

  if (use_vector) {
    using vec_t = flashinfer::vec_t<T, vec_size>;
    const int32_t num_vec_elems = hidden_dim / vec_size;

    for (int32_t i = tid; i < num_vec_elems; i += block_dim) {
      vec_t input_vec;
      input_vec.cast_load(token_input + i * vec_size);
      const int32_t base_idx = i * vec_size;

      FP8_TYPE output_arr[vec_size];
#pragma unroll
      for (uint32_t j = 0; j < vec_size; ++j) {
        float val = static_cast<float>(input_vec[j]) * scale_val;
        val = fmaxf(fminf(val, FP8_E4M3_MAX), -FP8_E4M3_MAX);
#ifndef USE_ROCM
        output_arr[j] = static_cast<FP8_TYPE>(val);
#else
        output_arr[j] = c10::Float8_e4m3fnuz(
            __hip_cvt_float_to_fp8(val, fp8::fp8_type::__default_saturation, fp8::fp8_type::__default_interpret),
            c10::Float8_e4m3fnuz::from_bits());
#endif
        token_output[base_idx + j] = output_arr[j];
      }
    }
  } else {
    for (int32_t idx = tid; idx < hidden_dim; idx += block_dim) {
      float val = static_cast<float>(token_input[idx]) * scale_val;
      val = fmaxf(fminf(val, FP8_E4M3_MAX), -FP8_E4M3_MAX);
#ifndef USE_ROCM
      token_output[idx] = static_cast<FP8_TYPE>(val);
#else
      token_output[idx] = c10::Float8_e4m3fnuz(
          __hip_cvt_float_to_fp8(val, fp8::fp8_type::__default_saturation, fp8::fp8_type::__default_interpret),
          c10::Float8_e4m3fnuz::from_bits());
#endif
    }
  }
}

void sgl_per_token_quant_fp8(torch::Tensor input, torch::Tensor output_q, torch::Tensor output_s) {
  CHECK_INPUT(input);
  CHECK_INPUT(output_q);
  CHECK_INPUT(output_s);

  const auto input_sizes = input.sizes();
  const int64_t num_tokens = input_sizes[0];
  const int64_t hidden_dim = input_sizes[1];

  const int block_size = 256;
  const int num_blocks = num_tokens;

  dim3 grid(num_blocks);
  dim3 block(block_size);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), scalar_t, [&] {
    per_token_quant_fp8_kernel<scalar_t><<<grid, block, 0, stream>>>(
        static_cast<scalar_t*>(input.data_ptr()),
        static_cast<FP8_TYPE*>(output_q.data_ptr()),
        static_cast<float*>(output_s.data_ptr()),
        hidden_dim,
        num_tokens);
    return true;
  });
}
